#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hello_world.hpp"

__global__ void helloFromGPU(){
  printf("Hello World from GPU\n");
}

void launch_cuda(){
  helloFromGPU<<<1, 10>>>();
  hipDeviceReset();
}
