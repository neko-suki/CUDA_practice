
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sumArraysOnGpu(const float *a, const float *b, float *c){
  const size_t i = threadIdx.x;
  
  c[i] = a[i] + b[i];
}

void launch_cuda(const size_t n, const size_t nBytes, const float * a, const float * b, float * c){
  float *d_A, *d_B, *d_C;
  hipMalloc((float**) &d_A, nBytes);
  hipMalloc((float**) &d_B, nBytes);
  hipMalloc((float**) &d_C, nBytes);
  
  hipMemcpy(d_A, a, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, b, nBytes, hipMemcpyHostToDevice);
  
  sumArraysOnGpu<<<1, n>>>(d_A, d_B, d_C);

  hipMemcpy(c, d_C, nBytes, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
